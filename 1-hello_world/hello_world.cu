
#include <hip/hip_runtime.h>
#include <stdio.h>

// 使用 __global__ 关键字定义 CUDA 内核函数（在 GPU 上执行）
// 此函数会被多个线程并行调用
__global__ void hello_world(void) { printf("GPU: Hello world!\n"); }

int main(int argc, char **argv) {
  printf("CPU: Hello world!\n");

  // 调用 CUDA 内核函数
  // 执行配置 <<<网格维度, 线程块维度>>>：此处启动 1 个线程块，每个块有 10
  // 个线程
  hello_world<<<1, 10>>>();

  // 清理 GPU 资源（重置当前设备，释放所有分配的内存）
  // 这句话包含了隐式同步，GPU和CPU执行程序是异步的，核函数调用后成立刻会到主机线程继续，而不管GPU端核函数是否执行完毕，所以上面的程序就是GPU刚开始执行，CPU已经退出程序了，所以我们要等GPU执行完了，再退出主机
  hipDeviceReset();
  return 0;
}